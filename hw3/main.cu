#include "hip/hip_runtime.h"
// driver_api.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cerrno>

int funcPrepare(const char* filePath, hipModule_t& module, hipFunction_t& function, const char* funcName, char*& cubin, hipError_t& err){
    // 加载CUBIN文件
    // 修改文件加载部分的代码
    FILE* file = fopen(filePath, "rb");
    if (!file) {  // 必须检查文件是否成功打开
        printf("Error: Cannot open cubin_data.cubin Reason: %s\n", strerror(errno));
        printf("Current working directory: ");
        system("pwd");  // 打印当前工作目录
        return -1;
    }

    fseek(file, 0, SEEK_END);
    size_t size = ftell(file);
    fseek(file, 0, SEEK_SET);
    cubin = (char*)malloc(size);
    size_t read_size = fread(cubin, 1, size, file);
    if (read_size != size) {
        printf("Error: Failed to read full file (expected %zu, got %zu)\n", size, read_size);
        return -1;
    }
    fclose(file);

    // 加载模块和函数
    if ((err = hipModuleLoadData(&module, cubin)) != hipSuccess) {
        printf("hipModuleLoadData failed: %d\n", err);
        return -1;
    }
    if ((err = hipModuleGetFunction(&function, module, funcName)) != hipSuccess) {
        printf("hipModuleGetFunction failed: %d \n %s \n", err, funcName);
        return -1;
    }
     return 0;  // Return success
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module_add;
    hipModule_t module_matmul;
    hipFunction_t function_add;
    hipFunction_t function_matmul;
    char * cubin;
    hipError_t err;

    // 初始化Driver API
    if ((err = hipInit(0)) != hipSuccess) {
        printf("hipInit failed: %d\n", err);
        return -1;
    }
    if ((err = hipDeviceGet(&device, 0)) != hipSuccess) {
        printf("hipDeviceGet failed: %d\n", err);
        return -1;
    }
    if ((err = hipCtxCreate(&context, 0, device)) != hipSuccess) {
        printf("hipCtxCreate failed: %d\n", err);
        return -1;
    }

    // 加载add_kernel
    if (funcPrepare("cubin_data.cubin", module_add, function_add, "add_kernel", cubin, err) < 0) {
        return -1;
    }

    // 加载naive_matmul_k kernel
    if (funcPrepare("naive_matmul_k.cubin", module_matmul, function_matmul, "naive_matmul_k", cubin, err) < 0) {
        return -1;
    }

    // 准备数据
    hipDeviceptr_t d_x, d_y, d_output;
    hipMalloc(&d_x, 1024 * sizeof(float));
    hipMalloc(&d_y, 1024 * sizeof(float));
    hipMalloc(&d_output, 1024 * sizeof(float));

    // 准备输入数据
    float h_x[1024], h_y[1024], h_output[1024];
    for (int i = 0; i < 1024; i++) {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }
    hipMemcpyHtoD(d_x, h_x, 1024 * sizeof(float));
    hipMemcpyHtoD(d_y, h_y, 1024 * sizeof(float));

    // 启动内核
    int len = 1024;
    void* args[] = { &d_x, &d_y, &d_output, &len };
    hipModuleLaunchKernel(function_add, 1, 1, 1, 1024, 1, 1, 0, NULL, args, NULL);

    // 验证结果
    hipMemcpyDtoH(h_output, d_output, 1024 * sizeof(float));
    bool passed = true;
    for (int i = 0; i < 1024; i++) {
        if (fabs(h_output[i] - 3.0f) > 1e-6) {  // 1 + 2 = 3
            printf("Verification failed at index %d: expected 3.0, got %f\n", i, h_output[i]);
            passed = false;
            break;
        }
    }
    if (passed) {
        printf("Kernel verification PASSED!\n");
    }

    // ====================== 测试 naive_matmul_k ======================
    // 准备矩阵乘法测试数据 (128x256) * (256x512) = (128x512)
    int M = 128, N = 512, K = 256;
    hipDeviceptr_t d_A, d_B, d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // 初始化随机输入数据
    float *h_A = (float*)malloc(M * K * sizeof(float));
    float *h_B = (float*)malloc(K * N * sizeof(float));
    float *h_C = (float*)malloc(M * N * sizeof(float));
    // 保持与Python相同的随机数范围(0.1倍缩放)
    for (int i = 0; i < M * K; i++) h_A[i] = (float)rand() / RAND_MAX * 0.1f;
    for (int i = 0; i < K * N; i++) h_B[i] = (float)rand() / RAND_MAX * 0.1f;
    
    hipMemcpyHtoD(d_A, h_A, M * K * sizeof(float));
    hipMemcpyHtoD(d_B, h_B, K * N * sizeof(float));

    // 定义block size常量
    const int bs = 16;
    int stride_am = K, stride_ak = 1;
    int stride_bk = N, stride_bn = 1;
    int stride_cm = N, stride_cn = 1;
    int bm = bs, bn = bs, bk = bs;

    // 设置kernel参数（正确传递设备指针）
    void* matmul_args[] = {
        &d_A,      // 直接传递设备指针值
        &d_B, 
        &d_C,
        &M, &N, &K,
        &stride_am, &stride_ak,
        &stride_bk, &stride_bn,
        &stride_cm, &stride_cn,
        &bm, &bn, &bk
    };

    // 启动kernel (使用正确的block配置)
    dim3 grid((M + bm - 1) / bm, (N + bn - 1) / bn);
    dim3 block(16, 16);  // 必须与Triton kernel的bm/bn一致
    // 添加共享内存配置（根据Triton kernel需求）
    const int shared_mem = 2 * bm * bk * sizeof(float) + 2 * bk * bn * sizeof(float);
    
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);
    
    hipModuleLaunchKernel(function_matmul, 
                  grid.x, grid.y, 1,    // grid dim
                  block.x, block.y, 1,  // block dim
                  shared_mem, stream, matmul_args, NULL);  // 添加共享内存配置
    
    // 等待kernel完成
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    // 验证结果
    hipMemcpyDtoH(h_C, d_C, M * N * sizeof(float));
    
    // 简单验证几个样本点
    bool matmul_passed = true;
    for (int i = 0; i < 10; i++) {
        int row = rand() % M;
        int col = rand() % N;
        float expected = 0.0f;
        for (int k = 0; k < K; k++) {
            expected += h_A[row * K + k] * h_B[k * N + col];
        }
        if (fabs(h_C[row * N + col] - expected) > 1e-3) {
            printf("Matmul verification failed at (%d,%d): expected %.4f, got %.4f, i=%d\n", 
                  row, col, expected, h_C[row * N + col], i);
            matmul_passed = false;
            break;
        }
    }
    if (matmul_passed) {
        printf("Matmul kernel verification PASSED!\n");
    }

    // 清理资源
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_output);
    hipModuleUnload(module_add);
    hipModuleUnload(module_matmul);
    hipCtxDestroy(context);
    free(cubin);
    return 0;
}